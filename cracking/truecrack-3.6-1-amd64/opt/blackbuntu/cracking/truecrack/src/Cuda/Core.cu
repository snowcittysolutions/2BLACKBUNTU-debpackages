#include "hip/hip_runtime.h"
/*
 * Copyright (C)  2011  Luca Vaccaro
 *
 * TrueCrack is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "Tcdefs.h"
#include "Volumes.cuh"
#include <stdio.h>
#include <memory.h>
#include "Crypto.cuh"
#include "Core.cuh"
#include "Pkcs5.cuh"
#include "Xts.cuh"

/* Constat gpu memory data */
__device__ __constant__ unsigned char cHeaderEncrypted[TC_VOLUME_HEADER_EFFECTIVE_SIZE];
__device__ __constant__ unsigned char cSalt[SALT_LENGTH];

/* Header key size */
#define MAXPKCS5OUTSIZE 64

/* The max number of block grid; number of max parallel gpu blocks. */
int blockGridSizeMax;

/* The number of the current block grid; number of current parallel gpu blocks. */
int blockGridSizeCurrent;


/* Pointer of structures to pass to Cuda Kernel. */
unsigned char *dev_salt, *dev_blockPwd, *dev_header, *dev_headerKey;
int *dev_blockPwd_init, *dev_blockPwd_length;
short int *dev_result;
/* With Stream
#define NSTREAM 6
unsigned char *dev_salt, *dev_blockPwd[NSTREAM], *dev_header, *dev_headerKey[NSTREAM];
int *dev_blockPwd_init[NSTREAM], *dev_blockPwd_length[NSTREAM];
short int *dev_result;
*/

int getMultiprocessorCount (void){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	return prop.multiProcessorCount;
}

//#define RESIDENTTHREADS		1536
//#define NUMBLOCKS		12
#define NUMTHREADSXBLOCK	256

// Handle GPU error
static void HandleError( hipError_t err, const char *file,  int line ) {
        if (err != hipSuccess) {
                printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line );
                exit( EXIT_FAILURE );
        }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


// GPU kernel: build word from an alphabet
__global__ void cuKernel_generate(unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, int offset, uint32_t maxsize, int charsetlength, unsigned char *charset, int wordlength){
    int number=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
    if (number>=maxsize) {blockPwd_init[number]=1;return;}
	
    blockPwd_init[number]=number*wordlength;//(number==0)?0:blockPwd_init[number-1]+wordlength;
    blockPwd_length[number]=wordlength;
    
    unsigned char *word; word= &blockPwd[number*wordlength];
    unsigned short i=0;
    for (i=0;i<wordlength;i++)
        word[i]=0;
    i=0;
    number+=offset;
    while(number>0){
        word[i]=number%charsetlength;
        number=(number-word[i])/charsetlength;
        i++;
    }
    
    for (i=0;i<wordlength;i++)
    	word[i]=charset[word[i]];
}

// GPU kernel: ripemd160 hash
__global__ void cuKernel_ripemd160 (unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, unsigned char *headerKey, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	cuda_derive_key_ripemd160 (  blockPwd+blockPwd_init[numData], blockPwd_length[numData], cSalt, PKCS5_SALT_SIZE, 2000, headerKey+numData*MAXPKCS5OUTSIZE, 64);
}
// GPU kernel: sha512 hash
__global__ void cuKernel_sha512 ( unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, unsigned char *headerKey, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	cuda_derive_key_sha512 (  blockPwd+blockPwd_init[numData], blockPwd_length[numData], cSalt, PKCS5_SALT_SIZE, 1000, headerKey+numData*MAXPKCS5OUTSIZE, 64);
}
// GPU kernel: whirlpool hash
__global__ void cuKernel_whirlpool ( unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, unsigned char *headerKey, int max) {
       int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
        if (numData>=max) return;
        cuda_derive_key_whirlpool (  blockPwd+blockPwd_init[numData], blockPwd_length[numData], cSalt, PKCS5_SALT_SIZE, 1000, headerKey+numData*MAXPKCS5OUTSIZE, 64);
}
// GPU kernel: aes xts decryption
__global__ void cuKernel_aes ( unsigned char *headerKey, short int *result, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	__align__(8) unsigned char headerDecrypted[512];
	result[numData]=cuXts (AES,cHeaderEncrypted, headerKey+numData*MAXPKCS5OUTSIZE,headerDecrypted);
}
// GPU kernel: serpent xts decryption
__global__ void cuKernel_serpent ( unsigned char *headerKey, short int *result, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	__align__(8) unsigned char headerDecrypted[512];
	result[numData]=cuXts (SERPENT,cHeaderEncrypted, headerKey+numData*MAXPKCS5OUTSIZE,headerDecrypted);
}
// GPU kernel: twofish xts decryption
__global__ void cuKernel_twofish (unsigned char *headerKey, short int *result, int max) {
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;
	if (numData>=max) return;
	__align__(8) unsigned char headerDecrypted[512];
	result[numData]=cuXts (TWOFISH,cHeaderEncrypted, headerKey+numData*MAXPKCS5OUTSIZE,headerDecrypted);
}

// Perform the bruteforce on dictionary mode
float cuda_Core_dictionary ( int encryptionAlgorithm, int bsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int keyDerivationFunction) {
	// Initialization
	int lengthpwd=0;
	for (int j=0;j<bsize;j++) {
		lengthpwd+=blockPwd_length[j];
		result[j]=0;
	}
	// Copy memory datas from host to gpu
	HANDLE_ERROR(hipMemcpy(dev_blockPwd, 		blockPwd, 		lengthpwd * sizeof(unsigned char) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_blockPwd_init, 	blockPwd_init, 	bsize * sizeof(int) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_blockPwd_length,blockPwd_length,bsize * sizeof(int) , hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_result, 		result,			bsize * sizeof(short int) , hipMemcpyHostToDevice));

	// Calculate the block grid and threads for block
	int numBlocks=bsize/NUMTHREADSXBLOCK+1;
	int numThreads=NUMTHREADSXBLOCK;
	if (bsize<NUMTHREADSXBLOCK)
		numThreads=bsize;

	// Start timer
	hipEvent_t tstart,tstop;
	float time;
	hipEventCreate(&tstart);
	hipEventCreate(&tstop);
	hipEventRecord(tstart, 0);

	// GPU Kernel: Key derivation function
	switch(keyDerivationFunction){
		case RIPEMD160:
			cuKernel_ripemd160 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey, bsize);
			break;
		case SHA512:
			cuKernel_sha512 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
		case WHIRLPOOL:
			cuKernel_whirlpool <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
	}
	
	// GPU Kernel: Encryption algorithms
	switch(encryptionAlgorithm){
		case AES:
			cuKernel_aes<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case SERPENT:
			cuKernel_serpent<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case TWOFISH:
			cuKernel_twofish<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
	}

	// Stop timer
	hipEventRecord(tstop, 0);
	hipEventSynchronize(tstop);
	hipEventElapsedTime(&time, tstart, tstop);

	// Copy memory result from gpu to host
	HANDLE_ERROR(hipMemcpy(result, dev_result,bsize* sizeof(short int) , hipMemcpyDeviceToHost));
	return time;
}


// Perform the bruteforce on charset mode
float cuda_Core_charset ( int encryptionAlgorithm, uint64_t bsize, uint64_t start, unsigned short int charset_length, unsigned char *charset, unsigned short int password_length, short int *result, int keyDerivationFunction)
{
	// Initialization
	int numBlocks=(int)(bsize/NUMTHREADSXBLOCK)+1;
	int numThreads=NUMTHREADSXBLOCK;
	if (bsize<NUMTHREADSXBLOCK)
		numThreads=(int)bsize;
		
	// Copy memory datas from host to gpu
	unsigned char *dev_charset = NULL;
	HANDLE_ERROR(hipMalloc((void **)&dev_charset, charset_length*sizeof(unsigned char)));
	HANDLE_ERROR(hipMemcpy(dev_charset, charset, charset_length*sizeof(unsigned char), hipMemcpyHostToDevice));
	/*
	char host_blockPwd[bsize*PASSWORD_MAXSIZE];
	int host_blockPwd_init[bsize];
	int host_blockPwd_length[bsize];
	*/
	// Start timer
    hipEvent_t tstart,tstop;
    float time;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);
    hipEventRecord(tstart, 0); 	

	// GPU Kernel: generate passwords
	cuKernel_generate <<<numBlocks,numThreads>>>(dev_blockPwd,dev_blockPwd_init,dev_blockPwd_length,(int)start,bsize,charset_length,dev_charset,password_length);
	
	// GPU Kernel: Key derivation function
	switch(keyDerivationFunction){
		case RIPEMD160:
			cuKernel_ripemd160 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey, bsize);
			break;
		case SHA512:
			cuKernel_sha512 <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
		case WHIRLPOOL:
			cuKernel_whirlpool <<<numBlocks,numThreads>>>(dev_blockPwd, dev_blockPwd_init, dev_blockPwd_length, dev_headerKey,bsize);
			break;
	}
	
	// GPU Kernel: Encryption algorithms
	switch(encryptionAlgorithm){
		case AES:
			cuKernel_aes<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case SERPENT:
			cuKernel_serpent<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
		case TWOFISH:
			cuKernel_twofish<<<numBlocks,numThreads>>>(dev_headerKey, dev_result, bsize);
			break;
	}
	
	// Stop timer
    hipEventRecord(tstop, 0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&time, tstart, tstop);
	/*
	HANDLE_ERROR( hipMemcpy(host_blockPwd, dev_blockPwd, bsize*PASSWORD_MAXSIZE*sizeof(unsigned char), hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipMemcpy(host_blockPwd_init, dev_blockPwd_init, bsize*sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR( hipMemcpy(host_blockPwd_length, dev_blockPwd_length, bsize*sizeof(int), hipMemcpyDeviceToHost));
	printf("host_blockPwd_init: ");
	for (int i=0;i<bsize;i++)
	  printf("%d",host_blockPwd_init[i]);
	printf("\nhost_blockPwd_length: ");
	for (int i=0;i<bsize;i++)
	  printf("%d",host_blockPwd_length[i]);
	printf("\nhost_blockPwd: ");	
	for (int i=0;i<bsize*PASSWORD_MAXSIZE;i++)
	  printf("%c",host_blockPwd[i]);
	printf("\n");
	*/
	// Copy memory result from gpu to host
	HANDLE_ERROR( hipMemcpy(result, dev_result, bsize*sizeof(short int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(dev_charset));
	return time;
}
   
// GPU memory initialization: malloc anc constant symbols
void cuda_Init (int bsize, unsigned char *salt, unsigned char *header) {
	HANDLE_ERROR(hipMalloc ( (void **)&dev_blockPwd, 		bsize*PASSWORD_MAXSIZE* sizeof(unsigned char))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_blockPwd_init,		bsize * sizeof(int))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_blockPwd_length, 	bsize * sizeof(int))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_headerKey, 		MAXPKCS5OUTSIZE * bsize * sizeof(unsigned char))) ;
	HANDLE_ERROR(hipMalloc ( (void **)&dev_result, 		bsize * sizeof(short int)))  ;
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( cSalt), 		salt , SALT_LENGTH* sizeof(unsigned char),0,hipMemcpyHostToDevice)) ;
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL( cHeaderEncrypted), 	header , TC_VOLUME_HEADER_EFFECTIVE_SIZE* sizeof(unsigned char),0,hipMemcpyHostToDevice)) ;
  
}

// GPU memory free and reset
void cuda_Free(void) {
	hipFree(dev_salt);
	hipFree(dev_blockPwd);
	hipFree(dev_blockPwd_init);
	hipFree(dev_blockPwd_length);
	hipFree(dev_result);
	hipFree(dev_headerKey);
	hipDeviceReset();
}


// LEGACY: 
/*
void cuda_Core_dictionary ( int block_currentsize, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int keyDerivationFunction) {

	int size_block=block_currentsize;
	int size_stream=block_currentsize/NSTREAM;
	
	int numBlocks=size_stream/NUMTHREADSXBLOCK+1;
	int numThread=NUMTHREADSXBLOCK;
	if (size_stream<NUMTHREADSXBLOCK)
		numThread=size_stream;

	hipStream_t stream[NSTREAM];
	for (int i = 0; i < NSTREAM; ++i)
		hipStreamCreate(&stream[i]);
	
	int lengthpwd[NSTREAM]={0};
	for (int i=0;i<NSTREAM;i++){
	  for (int j=0;j<size_stream;j++) {
		lengthpwd[i]+=blockPwd_length[j+i*size_stream];
	  }
	}
	printf("1-%d 2-%d \n",lengthpwd[0],lengthpwd[1]);
	
	hipMalloc ( &dev_result, size_block* sizeof(short int)) ;
	hipMemcpy ( dev_result, result, size_block* sizeof(short int),hipMemcpyHostToDevice);
	
	unsigned char *host_blockPwd[NSTREAM];
	int *host_blockPwd_init[NSTREAM];
	int *host_blockPwd_length[NSTREAM];
	short int *host_result[NSTREAM];
	
	for (int i =0; i<NSTREAM; i++){
	
		hipMalloc ( (void **)&dev_blockPwd[i], 	size_stream * PASSWORD_MAXSIZE * sizeof(unsigned char)) ;
		hipMalloc ( (void **)&dev_blockPwd_init[i], 	size_stream * sizeof(int)) ;
		hipMalloc ( (void **)&dev_blockPwd_length[i], 	size_stream * sizeof(int)) ;
		hipMalloc ( (void **)&dev_headerKey[i], 	256 * size_stream * sizeof(unsigned char)) ;
	
	        hipHostAlloc(&host_blockPwd[i], 	lengthpwd[i]* sizeof(unsigned char), 	hipHostMallocDefault);
		hipHostAlloc(&host_blockPwd_init[i], 	size_stream * sizeof(int), 		hipHostMallocDefault);
		hipHostAlloc(&host_blockPwd_length[i], size_stream * sizeof(int),	 	hipHostMallocDefault);
		hipHostAlloc(&host_result[i], 		size_stream * sizeof(int),	 	hipHostMallocDefault);
	
		memcpy(host_blockPwd[i], 	blockPwd+((i==0)?0:lengthpwd[i-1]),	lengthpwd[i]*sizeof(unsigned char));
		memcpy(host_blockPwd_init[i], 	blockPwd_init+i*size_stream, 		size_stream*sizeof(int));
		memcpy(host_blockPwd_length[i], blockPwd_length+i*size_stream, 		size_stream*sizeof(int));
			
	}
	
	for (int i = 0; i < NSTREAM; i++){
	  
		hipMemcpyAsync(dev_blockPwd[i], 	host_blockPwd[i],		lengthpwd[i] * sizeof(unsigned char) , hipMemcpyHostToDevice, stream[i]) ;
		hipMemcpyAsync(dev_blockPwd_init[i], 	host_blockPwd_init[i], 		size_stream * sizeof(int) , hipMemcpyHostToDevice,stream[i]);
		hipMemcpyAsync(dev_blockPwd_length[i],	host_blockPwd_length[i], 	size_stream * sizeof(int) , hipMemcpyHostToDevice,stream[i]) ;
		hipMemcpyAsync(dev_result, 		host_result[0], 		size_stream * sizeof(short int) , hipMemcpyHostToDevice,stream[0]) ;
		
		
		cuda_Kernel_ripemd160<<<numBlocks,numThread, 0, stream[i]>>>(dev_blockPwd[i], dev_blockPwd_init[i], dev_blockPwd_length[i], dev_headerKey[i], size_stream);
		cuda_Kernel_aes<<<numBlocks,numThread, 0, stream[i]>>>(dev_headerKey[i], dev_result, size_stream);
			
		hipError_t err=hipMemcpy(result+i*size_stream, dev_result,	size_stream* sizeof(short int) , hipMemcpyDeviceToHost) ;
	//	hipError_t err=hipMemcpyAsync(host_result[0], dev_result,	size_stream* sizeof(short int) , hipMemcpyDeviceToHost,stream[0]) ;
		if (err!=hipSuccess){
			printf("->%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
		}printf("ok %d\n",i);
		hipDeviceSynchronize();
//	memcpy(result, 	host_result[0], 		size_stream*sizeof(int));
	
	
		
		//cuda_Kernel_ripemd160<<<numBlocks,numThread, 0, stream[i]>>>(dev_blockPwd+ i * size_stream, dev_blockPwd_init+ i * size_stream, dev_blockPwd_length+ i * size_stream, dev_headerKey, size_stream);
		//cuda_Kernel_aes<<<numBlocks,numThread, 0, stream[i]>>>(dev_headerKey, dev_result+ i * size_stream, size_stream);
		
		
		//hipError_t err=hipMemcpyAsync(host_result+i*size_stream, 	dev_result+i*size_stream,	size_stream* sizeof(short int) , hipMemcpyDeviceToHost, stream[i]) ;
		
	}
	
	for (int i = 0; i < NSTREAM; i++)
		hipStreamDestroy(stream[i]);
    
    
	hipFree(dev_result);
}
*/

/*
 __global__ void cuda_Kernel_charset (
    	unsigned char *salt,
    	unsigned char *headerEncrypted,
    	unsigned short int charset_length,
    	unsigned char *charset,
    	unsigned short int password_length,
    	uint64_t maxcombination,
    	 short int *result, 
	 int keyDerivationFunction)
 {
	uint64_t numData = blockIdx.x*blockDim.x+threadIdx.x;
	__align__(8) unsigned char headerkey[192];
	__align__(8) unsigned char headerDecrypted[512];
	__align__(8) unsigned char pwd[8];

	//__device__ void computePwd (int number, int maxcombination, int charsetlength, unsigned char *charset, int wordlength, unsigned char *word){
	computePwd(numData,maxcombination,charset_length,charset,password_length,pwd);
	pwd[password_length]='\0';
	
	//__device__ void cuda_Pbkdf2_charset_ ( unsigned char *salt, unsigned char *pwd, int pwd_len, unsigned char *headerkey) {
//	cuda_Pbkdf2 ( salt, pwd, password_length, headerkey);

	int value=cuda_Xts (headerEncrypted, headerkey, headerDecrypted);
	if (value==SUCCESS)
		result[numData]=MATCH;
	else
		result[numData]=NOMATCH;
}*/

/*	
__global__ void cuda_Kernel ( unsigned char *salt, unsigned char *headerEncrypted, unsigned char *blockPwd, int *blockPwd_init, int *blockPwd_length, short int *result, int max, int keyDerivationFunction) {
	int value;
	int numData=blockIdx.x*NUMTHREADSXBLOCK+threadIdx.x;

	if (numData>=max) return;

	// Array of unsigned char in the shared memory
	__align__(8) unsigned char headerKey[192];
	__align__(8) unsigned char headerDecrypted[512];

	// Calculate the hash header key
	unsigned char *pwd=blockPwd+blockPwd_init[numData];
	int pwd_len = blockPwd_length[numData];


	if(keyDerivationFunction==RIPEMD160)
		cuda_Pbkdf2 ( salt, pwd, pwd_len, headerKey);
	else if(keyDerivationFunction==SHA512)
		cuda_derive_key_sha512 (  pwd, pwd_len, salt, PKCS5_SALT_SIZE, 1000, headerKey, 64);
	else if(keyDerivationFunction==WHIRLPOOL)
		cuda_derive_key_whirlpool (  pwd, pwd_len, salt, PKCS5_SALT_SIZE, 1000, headerKey, 64);
	else
		;
	
	// Decrypt the header and compare the key
	value=cuda_Xts (headerEncrypted, headerKey,headerDecrypted);

	if (value==SUCCESS)
		result[numData]=MATCH;
	else
		result[numData]=NOMATCH;
}
*/
